
#include <hip/hip_runtime.h>
/* -*- mode: c++ -*- */

#define UNROLL9(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\
	F(8);					\


#define UNROLL8x3x3(F)				\
	F(0,0,0);				\
	F(0,0,1);				\
	F(0,0,2);				\
	F(0,1,0);				\
	F(0,1,1);				\
	F(0,1,2);				\
	F(0,2,0);				\
	F(0,2,1);				\
	F(0,2,2);				\
						\
	F(1,0,0);				\
	F(1,0,1);				\
	F(1,0,2);				\
	F(1,1,0);				\
	F(1,1,1);				\
	F(1,1,2);				\
	F(1,2,0);				\
	F(1,2,1);				\
	F(1,2,2);				\
						\
	F(2,0,0);				\
	F(2,0,1);				\
	F(2,0,2);				\
	F(2,1,0);				\
	F(2,1,1);				\
	F(2,1,2);				\
	F(2,2,0);				\
	F(2,2,1);				\
	F(2,2,2);				\
						\
	F(3,0,0);				\
	F(3,0,1);				\
	F(3,0,2);				\
	F(3,1,0);				\
	F(3,1,1);				\
	F(3,1,2);				\
	F(3,2,0);				\
	F(3,2,1);				\
	F(3,2,2);				\
						\
	F(4,0,0);				\
	F(4,0,1);				\
	F(4,0,2);				\
	F(4,1,0);				\
	F(4,1,1);				\
	F(4,1,2);				\
	F(4,2,0);				\
	F(4,2,1);				\
	F(4,2,2);				\
						\
	F(5,0,0);				\
	F(5,0,1);				\
	F(5,0,2);				\
	F(5,1,0);				\
	F(5,1,1);				\
	F(5,1,2);				\
	F(5,2,0);				\
	F(5,2,1);				\
	F(5,2,2);				\
						\
	F(6,0,0);				\
	F(6,0,1);				\
	F(6,0,2);				\
	F(6,1,0);				\
	F(6,1,1);				\
	F(6,1,2);				\
	F(6,2,0);				\
	F(6,2,1);				\
	F(6,2,2);				\
						\
	F(7,0,0);				\
	F(7,0,1);				\
	F(7,0,2);				\
	F(7,1,0);				\
	F(7,1,1);				\
	F(7,1,2);				\
	F(7,2,0);				\
	F(7,2,1);				\
	F(7,2,2);				\

#define UNROLL8(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\


#define UNROLL8x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\


#define UNROLL10x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
	F(0,8);					\
	F(0,9);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
	F(1,8);					\
	F(1,9);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\
	F(2,8);					\
	F(2,9);					\


#define BLOCK_SIZE 8

template <int nInputPlanes>
__device__ void
filter(const float * __restrict__ packed_input,
       float * __restrict__ packed_output,
       int nOutputPlanes,
       const float * __restrict__ biases,
       unsigned int hsz,
       unsigned int wsz,
       const float * __restrict__ weight)
{
	extern __shared__ float shared_buf[];

	unsigned int yi = blockIdx.x;

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == hsz-1) {
		in2p = in1p;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	float *shared_ptr = shared_buf;
	float *in_block0_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block1_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block2_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);

	float *in_block0 = in_block0_base + nInputPlanes;
	float *in_block1 = in_block1_base + nInputPlanes;
	float *in_block2 = in_block2_base + nInputPlanes;
	int lid = threadIdx.x;
	float bv = biases[lid];

	for (int xi0=0; xi0<wsz; xi0+=BLOCK_SIZE) {

		/*for (unsigned int op=0; op<nOutputPlanes; op++) thread */
		{
			int op = lid;
			int rem = wsz - xi0;
			__syncthreads();
			if (lid < nInputPlanes/2) {
				int bi;
				int lid2 = lid*2;
				for (bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0 + bi;
					if (xi == wsz) {
						break;
					}

					/* load to shared */
					*(float2*)&in_block0[bi*nInputPlanes + lid2] = *(float2*)&in01[xi*nInputPlanes + lid2];
					*(float2*)&in_block1[bi*nInputPlanes + lid2] = *(float2*)&in11[xi*nInputPlanes + lid2];
					*(float2*)&in_block2[bi*nInputPlanes + lid2] = *(float2*)&in21[xi*nInputPlanes + lid2];
				}

				{
					int xi = xi0 + bi;
					if (xi == wsz) {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[(xi-1)*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[(xi-1)*(int)nInputPlanes + lid2];
					} else {
						*(float2*)&in_block0[bi*(int)nInputPlanes + lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[bi*(int)nInputPlanes + lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[bi*(int)nInputPlanes + lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}

				{
					int xi = xi0-1;
					if (xi == -1) {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[lid2];
					} else {
						*(float2*)&in_block0[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in01[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block1[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in11[xi*(int)nInputPlanes + lid2];
						*(float2*)&in_block2[-1*(int)nInputPlanes + (int)lid2] = *(float2*)&in21[xi*(int)nInputPlanes + lid2];
					}
				}
			}
			__syncthreads();

			if (rem >= BLOCK_SIZE) {
#define DECL_PTR(y,x)		float *p##y##x = &in_block##y[nInputPlanes * (x-1)];

				UNROLL10x3(DECL_PTR);

				float sum0 = 0;
				float sum1 = 0;
				float sum2 = 0;
				float sum3 = 0;

				float sum4 = 0;
				float sum5 = 0;
				float sum6 = 0;
				float sum7 = 0;

				{
					const float *w0 = weight + lid;

					for (int ip = 0; ip < nInputPlanes; ip++) {
#define LOAD_INPUT2(y,x)			float2 i##y##x##_2 = *(float2*)&p##y##x[ip];

						UNROLL10x3(LOAD_INPUT2);

#define LOAD_COEF(X)				float w_##X = w[X * 128];

#define CALC(IDX,Y,I0,I1,I2,I3,I4,I5,I6,I7)				\
						sum0 += w_##IDX * i##Y##I0; \
						sum1 += w_##IDX * i##Y##I1; \
						sum2 += w_##IDX * i##Y##I2; \
						sum3 += w_##IDX * i##Y##I3; \
						sum4 += w_##IDX * i##Y##I4; \
						sum5 += w_##IDX * i##Y##I5; \
						sum6 += w_##IDX * i##Y##I6; \
						sum7 += w_##IDX * i##Y##I7;


						{
#define LOAD_INPUT1X(Y,X)				float i##Y##X = i##Y##X##_2.x;

							UNROLL10x3(LOAD_INPUT1X);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

						ip++;
						{
#define LOAD_INPUT1Y(Y,X)				float i##Y##X = i##Y##X##_2.y;

							UNROLL10x3(LOAD_INPUT1Y);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

					}

#define RELU(BI)							\
					{				\
						float *out = packed_output + (yi*wsz + (xi0+BI))*nOutputPlanes; \
									\
						{			\
							int opIndex = lid; \
							float v = sum##BI; \
							v += bv;	\
									\
							float mtz = max(v, 0.0f); \
							float ltz = min(v, 0.0f); \
									\
							v = ltz * 0.1f + mtz; \
									\
							out[opIndex] = v; \
						}			\
					}

					UNROLL8(RELU);
				}
			} else {
				for (int bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0+bi;
					if (xi == wsz) {
						break;
					}

					const float *w0 = weight + lid;
					float sum = 0;

					for (int ip=0; ip<nInputPlanes; ip++) {
						float i00, i01, i02;
						float i10, i11, i12;
						float i20, i21, i22;

						i00 = in_block0[(bi-1)*nInputPlanes+ip];
						i10 = in_block1[(bi-1)*nInputPlanes+ip];
						i20 = in_block2[(bi-1)*nInputPlanes+ip];

						i01 = in_block0[bi*nInputPlanes+ip];
						i11 = in_block1[bi*nInputPlanes+ip];
						i21 = in_block2[bi*nInputPlanes+ip];

						i02 = in_block0[(bi+1)*nInputPlanes+ip];
						i12 = in_block1[(bi+1)*nInputPlanes+ip];
						i22 = in_block2[(bi+1)*nInputPlanes+ip];

						const float *w = w0;
						sum += w[(9*ip+0) * 128]*i00;
						sum += w[(9*ip+1) * 128]*i01;
						sum += w[(9*ip+2) * 128]*i02;

						sum += w[(9*ip+3) * 128]*i10;
						sum += w[(9*ip+4) * 128]*i11;
						sum += w[(9*ip+5) * 128]*i12;

						sum += w[(9*ip+6) * 128]*i20;
						sum += w[(9*ip+7) * 128]*i21;
						sum += w[(9*ip+8) * 128]*i22;
					}

					float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
					{
						float v = sum;
						v += bv;

						float mtz = max(v, 0.0f);
						float ltz = min(v, 0.0f);

						v = ltz * 0.1f + mtz;
						out[op] = v;
					}
				}
			}
		}
	}
}

extern "C" __global__ void
filter_i32(const float * __restrict__ packed_input,
	   float * __restrict__ packed_output,
	   int nOutputPlanes,
	   const float * __restrict__ biases,
	   unsigned int hsz,
	   unsigned int wsz,
	   const float * __restrict__ weight)
{
	filter<32>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

extern "C" __global__ void
filter_i64(const float * __restrict__ packed_input,
	   float * __restrict__ packed_output,
	   int nOutputPlanes,
	   const float * __restrict__ biases,
	   unsigned int hsz,
	   unsigned int wsz,
	   const float * __restrict__ weight)
{
	filter<64>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

extern "C" __global__ void
filter_i128(const float * __restrict__ packed_input,
	    float * __restrict__ packed_output,
	    int nOutputPlanes,
	    const float * __restrict__ biases,
	    unsigned int hsz,
	    unsigned int wsz,
	    const float * __restrict__ weight)
{
	filter<128>(packed_input, packed_output, nOutputPlanes, biases, hsz, wsz, weight);
}

static __device__ float
warp_sum(float v) {
    v += __shfl_down(v, 1);
    v += __shfl_down(v, 2);
    v += __shfl_down(v, 4);
    v += __shfl_down(v, 8);
    v += __shfl_down(v, 16);

    return v;
}

extern "C" __global__ void
filter_i128_o128(const float * __restrict__ packed_input,
		 float * __restrict__ packed_output,
		 const float * __restrict__ biases,
		 unsigned int hsz,
		 unsigned int wsz,
		 const float * __restrict__ weight)
{
	int nInputPlanes = 128;
	int nOutputPlanes = 128;

	/* 1024 thread
	 *  128 input plane x 32 output plane / block  (147KB regs)
	 *   4  input plane                   / thread (36 regs), 1output plane = 32thread
	 *
	 * block  [yi       , op32(0-3) ]
	 * thread [op1(0-31), ip(0-31)  ] (1024thread)
	 *
	 * op       = op32*32 + op1
	 * ip       = (ip*4+0, ip*4+1, ip*4+2, ip*4+3)
	 */
	int yi = blockIdx.y;
	int op32 = blockIdx.x;
	int op1 = threadIdx.y;
	int ip0 = threadIdx.x*32;

	int op = op32 * 32 + op1;

	float w000 = weight[(ip0*128 + op)*9 + 0];
	float w001 = weight[(ip0*128 + op)*9 + 1];
	float w002 = weight[(ip0*128 + op)*9 + 2];
	float w010 = weight[(ip0*128 + op)*9 + 3];
	float w011 = weight[(ip0*128 + op)*9 + 4];
	float w012 = weight[(ip0*128 + op)*9 + 5];
	float w020 = weight[(ip0*128 + op)*9 + 6];
	float w021 = weight[(ip0*128 + op)*9 + 7];
	float w022 = weight[(ip0*128 + op)*9 + 8];

	float w100 = weight[((ip0+1)*128 + op)*9 + 0];
	float w101 = weight[((ip0+1)*128 + op)*9 + 1];
	float w102 = weight[((ip0+1)*128 + op)*9 + 2];
	float w110 = weight[((ip0+1)*128 + op)*9 + 3];
	float w111 = weight[((ip0+1)*128 + op)*9 + 4];
	float w112 = weight[((ip0+1)*128 + op)*9 + 5];
	float w120 = weight[((ip0+1)*128 + op)*9 + 6];
	float w121 = weight[((ip0+1)*128 + op)*9 + 7];
	float w122 = weight[((ip0+1)*128 + op)*9 + 8];

	float w200 = weight[((ip0+2)*128 + op)*9 + 0];
	float w201 = weight[((ip0+2)*128 + op)*9 + 1];
	float w202 = weight[((ip0+2)*128 + op)*9 + 2];
	float w210 = weight[((ip0+2)*128 + op)*9 + 3];
	float w211 = weight[((ip0+2)*128 + op)*9 + 4];
	float w212 = weight[((ip0+2)*128 + op)*9 + 5];
	float w220 = weight[((ip0+2)*128 + op)*9 + 6];
	float w221 = weight[((ip0+2)*128 + op)*9 + 7];
	float w222 = weight[((ip0+2)*128 + op)*9 + 8];

	float w300 = weight[((ip0+3)*128 + op)*9 + 0];
	float w301 = weight[((ip0+3)*128 + op)*9 + 1];
	float w302 = weight[((ip0+3)*128 + op)*9 + 2];
	float w310 = weight[((ip0+3)*128 + op)*9 + 3];
	float w311 = weight[((ip0+3)*128 + op)*9 + 4];
	float w312 = weight[((ip0+3)*128 + op)*9 + 5];
	float w320 = weight[((ip0+3)*128 + op)*9 + 6];
	float w321 = weight[((ip0+3)*128 + op)*9 + 7];
	float w322 = weight[((ip0+3)*128 + op)*9 + 8];

	__shared__ float intermediate0[1024];
	__shared__ float intermediate1[1024];
	__shared__ float intermediate2[1024];
	__shared__ float intermediate3[1024];

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == hsz-1) {
		in2p = in1p;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	for (int xi=0; xi<wsz; xi++) {
		float sum = 0;
#define CONVOLVE(I) {							\
			float v00, v01, v02;				\
			float v10, v11, v12;				\
			float v20, v21, v22;				\
									\
			v01 = in0p[xi*nInputPlanes + ip0 + I];		\
			v11 = in0p[xi*nInputPlanes + ip0 + I];		\
			v21 = in0p[xi*nInputPlanes + ip0 + I];		\
									\
			if (xi == 0) {					\
				v00 = v01;				\
				v10 = v11;				\
				v20 = v21;				\
			} else {					\
				v01 = in0p[(xi-1)*nInputPlanes + ip0 + I]; \
				v11 = in0p[(xi-1)*nInputPlanes + ip0 + I]; \
				v21 = in0p[(xi-1)*nInputPlanes + ip0 + I]; \
			}						\
									\
			if (xi == wsz-1) {				\
				v02 = v01;				\
				v12 = v11;				\
				v22 = v21;				\
			} else {					\
				v02 = in0p[(xi+1)*nInputPlanes + ip0 + I]; \
				v12 = in0p[(xi+1)*nInputPlanes + ip0 + I]; \
				v22 = in0p[(xi+1)*nInputPlanes + ip0 + I]; \
			}						\
									\
			sum += w##I##00 * v00;				\
			sum += w##I##01 * v01;				\
			sum += w##I##02 * v02;				\
									\
			sum += w##I##10 * v10;				\
			sum += w##I##11 * v11;				\
			sum += w##I##12 * v12;				\
									\
			sum += w##I##20 * v20;				\
			sum += w##I##21 * v21;				\
			sum += w##I##22 * v22;				\
		}

		CONVOLVE(0);
		CONVOLVE(1);
		CONVOLVE(2);
		CONVOLVE(3);

		sum = warp_sum(sum);

		if (ip0 == 0) {
			float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
			out[op] = sum;
		}
	}
}
